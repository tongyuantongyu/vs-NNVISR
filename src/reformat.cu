#include "hip/hip_runtime.h"
#include "reformat.h"
#include <hip/hip_fp16.h>
#include <type_traits>

half __device__ round(half f) {
  const half v0_5 = float(0.5);
  return hfloor(f + v0_5);
}

template<class F, class U>
static void __global__ fma_from(md_view<F, 2> dst, md_view<const U, 2> src, F a, F b) {
  uint32_t dst_x = threadIdx.x + blockDim.x * blockIdx.x;
  uint32_t dst_y = threadIdx.y + blockDim.y * blockIdx.y;

  auto [dst_h, dst_w] = dst.shape;
  if (dst_x >= dst_w || dst_y >= dst_h) {
    return;
  }

  auto [src_h, src_w] = src.shape;
  uint32_t src_x = dst_x >= src_w ? src_w - 1 : dst_x;
  uint32_t src_y = dst_y >= src_h ? src_h - 1 : dst_y;

  F value = static_cast<F>(src.at(src_y, src_x));
  value = a * value + b;
  dst.at(dst_y, dst_x) = value;
}

template<class F, class U>
static void __global__ fma_to(md_view<U, 2> dst, md_view<const F, 2> src, F a, F b, F min, F max) {
  uint32_t dst_x = threadIdx.x + blockDim.x * blockIdx.x;
  uint32_t dst_y = threadIdx.y + blockDim.y * blockIdx.y;

  auto [dst_h, dst_w] = dst.shape;
  if (dst_x >= dst_w || dst_y >= dst_h) {
    return;
  }

  F value = static_cast<F>(src.at(dst_y, dst_x));
  value = a * value + b;
  if constexpr (std::is_integral_v<U>) {
    value = round(value);
  }

  if (value < min) {
    value = min;
  }
  else if (value > max) {
    value = max;
  }

  if constexpr (std::is_integral_v<U> && sizeof(U) == 1) {
    dst.at(dst_y, dst_x) = static_cast<U>(static_cast<int16_t>(value));
  } else {
    dst.at(dst_y, dst_x) = static_cast<U>(value);
  }
}

template<class F, class U>
void import_pixel(md_view<F, 2> dst, md_view<const U, 2> src, float a, float b, hipStream_t stream) {
  dim3 dimBlock(32, 32);
  dim3 dimGrid;
  auto [dst_h, dst_w] = dst.shape;
  dimGrid.x = (dst_w + 31) / 32;
  dimGrid.y = (dst_h + 31) / 32;

  fma_from<<<dimGrid, dimBlock, 0, stream>>>(dst, src, F(a), F(b));
}

template void import_pixel<float, uint8_t>(md_view<float, 2> dst, md_view<const uint8_t, 2> src, float a, float b,
                                           hipStream_t stream);
template void import_pixel<half, uint8_t>(md_view<half, 2> dst, md_view<const uint8_t, 2> src, float a, float b,
                                          hipStream_t stream);
template void import_pixel<float, uint16_t>(md_view<float, 2> dst, md_view<const uint16_t, 2> src, float a, float b,
                                            hipStream_t stream);
template void import_pixel<half, uint16_t>(md_view<half, 2> dst, md_view<const uint16_t, 2> src, float a, float b,
                                           hipStream_t stream);
template void import_pixel<float, half>(md_view<float, 2> dst, md_view<const half, 2> src, float a, float b,
                                        hipStream_t stream);
template void import_pixel<half, half>(md_view<half, 2> dst, md_view<const half, 2> src, float a, float b,
                                       hipStream_t stream);
template void import_pixel<float, float>(md_view<float, 2> dst, md_view<const float, 2> src, float a, float b,
                                         hipStream_t stream);
template void import_pixel<half, float>(md_view<half, 2> dst, md_view<const float, 2> src, float a, float b,
                                        hipStream_t stream);

template<class F, class U>
void export_pixel(md_view<U, 2> dst, md_view<const F, 2> src, float a, float b, float min, float max, hipStream_t stream) {
  dim3 dimBlock(32, 32);
  dim3 dimGrid;
  auto [dst_h, dst_w] = dst.shape;
  dimGrid.x = (dst_w + 31) / 32;
  dimGrid.y = (dst_h + 31) / 32;

  fma_to<<<dimGrid, dimBlock, 0, stream>>>(dst, src, F(a), F(b), F(min), F(max));
}

template void export_pixel<float, uint8_t>(md_view<uint8_t, 2> dst, md_view<const float, 2> src, float a, float b, float min,
                                    float max, hipStream_t stream);
template void export_pixel<half, uint8_t>(md_view<uint8_t, 2> dst, md_view<const half, 2> src, float a, float b, float min,
                                   float max, hipStream_t stream);
template void export_pixel<float, uint16_t>(md_view<uint16_t, 2> dst, md_view<const float, 2> src, float a, float b, float min,
                                     float max, hipStream_t stream);
template void export_pixel<half, uint16_t>(md_view<uint16_t, 2> dst, md_view<const half, 2> src, float a, float b, float min,
                                    float max, hipStream_t stream);
template void export_pixel<float, half>(md_view<half, 2> dst, md_view<const float, 2> src, float a, float b, float min,
                                    float max, hipStream_t stream);
template void export_pixel<half, half>(md_view<half, 2> dst, md_view<const half, 2> src, float a, float b, float min,
                                   float max, hipStream_t stream);
template void export_pixel<float, float>(md_view<float, 2> dst, md_view<const float, 2> src, float a, float b, float min,
                                     float max, hipStream_t stream);
template void export_pixel<half, float>(md_view<float, 2> dst, md_view<const half, 2> src, float a, float b, float min,
                                    float max, hipStream_t stream);